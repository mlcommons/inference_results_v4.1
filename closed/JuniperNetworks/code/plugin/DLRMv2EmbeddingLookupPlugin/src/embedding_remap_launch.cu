/*
 * Copyright (c) 2024, NVIDIA CORPORATION. All rights reserved.
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
#include <cstdint>
#include <cstdio>
#include <iostream>

#include <hip/hip_fp16.h>
#include <cuda_pipeline_primitives.h>

#include "dlrmv2Helper.h"
#include "embedding_remap_kernel.cuh"
#include "embedding_remap_launch.h"

void remapEmbeddingRows(hipStream_t stream, const void* srcEmbeddings, void* dstEmbeddings, const int* newLocations,
    const unsigned int embeddingSize, const unsigned int embeddingRows, const unsigned int maxEmbeddingRowsOnGpu,
    const unsigned int embed_elem_size)
{
    if (embed_elem_size == sizeof(float))
        remapEmbeddingRows<float><<<embeddingRows, embeddingSize, 0, stream>>>(
            srcEmbeddings, dstEmbeddings, newLocations, embeddingSize, embeddingRows, maxEmbeddingRowsOnGpu);

    else if (embed_elem_size == sizeof(half))
        remapEmbeddingRows<half><<<embeddingRows, embeddingSize, 0, stream>>>(
            srcEmbeddings, dstEmbeddings, newLocations, embeddingSize, embeddingRows, maxEmbeddingRowsOnGpu);

    else if (embed_elem_size == sizeof(int8_t))
        remapEmbeddingRows<int8_t><<<embeddingRows, embeddingSize, 0, stream>>>(
            srcEmbeddings, dstEmbeddings, newLocations, embeddingSize, embeddingRows, maxEmbeddingRowsOnGpu);

    else ASSERT(false);

    CUDA_ASSERT(hipGetLastError());
}
